#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "cpu_calculations.h"

//========================================================================================================

void gpu_time()
{
    static hipEvent_t start, stop;

    if (start == NULL) //Start measuring
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
    }
    else //Stop measuring
    {
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("Total time for GPU calculations: %.03lfs\n\n", milliseconds/1000);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        start = stop = NULL;
    }
}

//========================================================================================================

__global__ void single_kernel_calculations(float* A, float* B, float* C, float* D, float* E, float* F, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && j < N)
    {
        float resAC = 0.0f, resBD = 0.0f, resAD = 0.0f, resBC = 0.0f;

        for (int k = 0; k < N; k++)
        {
            resAC += A[i * N + k] * C[k * N + j];
            resBD += B[i * N + k] * D[k * N + j];
            resAD += A[i * N + k] * D[k * N + j];
            resBC += B[i * N + k] * C[k * N + j];
        }

        E[i*N + j] = resAC - resBD;
        F[i*N + j] = resAD + resBC;
    }
}

//========================================================================================================

__global__ void multiply_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("(%d, %d)\n", i, j);

    if (i < N && j < N)
    {
        for (int k = 0; k < N; k++)
            R[i * N + j] += M1[i * N + k] * M2[k * N + j];
    }
}

//========================================================================================================

__global__ void add_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] + M2[pos];
    }
}

//========================================================================================================

__global__ void sub_matrix(float *R, float *M1, float *M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] - M2[pos];
    }
}

//========================================================================================================

int main(int argc, char **argv)
{
    int single_kernel_mode = 0;

    if (argc == 1)
    {
        printf("Give matrix size\n");
        return 0;
    }

    if (argc == 3)
    {
        //Mode 0 -> Multiple kernels
        //Mode 1 -> Single kernel
        //Mode 2 -> Run both

        if (strcmp(argv[2], "0") != 0 && strcmp(argv[2], "1") != 0 && strcmp(argv[2], "2") != 0)
        {
            printf("Invalid mode %s\n", argv[2]);
            return 0;
        }

        single_kernel_mode = atoi(argv[2]);
    }

    int N = atoi(argv[1]);

    // Initialize matrices in host
    //===========================================================================
    float *A, *B, *C, *D;
    initialize_matrices(&A, &B, &C, &D, N);

    int arraySize = N * N * sizeof(float);

    int blockSize = 16;
    dim3 block(blockSize, blockSize); // 16x16 = 256 threads per block. A multiple of 32, the warp size
    dim3 grid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    //Start CPU calculations
    //===========================================================================
    float *Ecpu, *Fcpu;

    posix_memalign((void**)&Ecpu, 32, arraySize);
    posix_memalign((void**)&Fcpu, 32, arraySize);

    cpu_calculation(A, B, C, D, N, Ecpu, Fcpu);

    //Initialize GPU memory
    //===========================================================================
    float *devA, *devB, *devC, *devD, *devE, *devF, *devAC, *devBD, *devAD, *devBC;

    hipMalloc(&devA, arraySize);
    hipMalloc(&devB, arraySize);
    hipMalloc(&devC, arraySize);
    hipMalloc(&devD, arraySize);
    hipMalloc(&devE, arraySize);
    hipMalloc(&devF, arraySize);

    hipMemcpy(devA, A, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devC, C, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, arraySize, hipMemcpyHostToDevice);

    if (single_kernel_mode != 1)
    {
        hipMalloc(&devAC, arraySize);
        hipMalloc(&devBD, arraySize);
        hipMalloc(&devAD, arraySize);
        hipMalloc(&devBC, arraySize);

        hipMemset(devAC, 0, arraySize);
        hipMemset(devBD, 0, arraySize);
        hipMemset(devAD, 0, arraySize);
        hipMemset(devBC, 0, arraySize);
    }

    //Start GPU calculations
    //===========================================================================
    float* E = (float *)malloc(arraySize);
    float* F = (float *)malloc(arraySize);

    printf("Performing GPU calculations...\n\n");

    //Start multiple kernel calculations
    if (single_kernel_mode != 1)
    {
        printf("> Running version with multiple kernels...\n-----------------------------------------------------\n");

        gpu_time();

        multiply_matrix<<<grid, block>>>(devAC, devA, devC, N);
        multiply_matrix<<<grid, block>>>(devBD, devB, devD, N);
        multiply_matrix<<<grid, block>>>(devAD, devA, devD, N);
        multiply_matrix<<<grid, block>>>(devBC, devB, devC, N);

        sub_matrix<<<grid, block>>>(devE, devAC, devBD, N);
        add_matrix<<<grid, block>>>(devF, devAD, devBC, N);

        gpu_time();

        hipFree(devAC);
        hipFree(devBD);
        hipFree(devAD);
        hipFree(devBC);

        //Compare results for multiple kernels
        hipMemcpy(E, devE, arraySize, hipMemcpyDeviceToHost);
        hipMemcpy(F, devF, arraySize, hipMemcpyDeviceToHost);
        matrix_comparison(Ecpu, Fcpu, E, F, N);
    }

    //Start single kernel calculations
    if (single_kernel_mode != 0)
    {
        printf("> Running version with single kernel...\n-----------------------------------------------------\n");

        gpu_time();
        single_kernel_calculations<<<grid, block>>>(devA, devB, devC, devD, devE, devF, N);
        gpu_time();

        //Compare results for single kernel
        hipMemcpy(E, devE, arraySize, hipMemcpyDeviceToHost);
        hipMemcpy(F, devF, arraySize, hipMemcpyDeviceToHost);
        matrix_comparison(Ecpu, Fcpu, E, F, N);
    }

    //Free memory
    //========================================================================================
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipFree(devD);

    free(A);
    free(B);
    free(C);
    free(D);
    free(E);
    free(F);
    free(Ecpu);
    free(Fcpu);
}