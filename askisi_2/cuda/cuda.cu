#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "cpu_calculations.h"

#include <sys/mman.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <unistd.h>

//========================================================================================================

double gpu_time()
{
    static hipEvent_t start, stop;

    if (start == NULL) //Start measuring
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);

        return 0;
    }
    else //Stop measuring
    {
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        printf("Total time for GPU calculations: %.03lfs\n\n", milliseconds/1000);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        start = stop = NULL;

        return milliseconds/1000.0;
    }
}

//========================================================================================================

// Calculations on GPU with one kernel
__global__ void single_kernel_calculations(float* A, float* B, float* C, float* D, float* E, float* F, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && j < N)
    {
        float resAC = 0.0f, resBD = 0.0f, resAD = 0.0f, resBC = 0.0f;

        for (int k = 0; k < N; k++)
        {
            resAC += A[i * N + k] * C[k * N + j];
            resBD += B[i * N + k] * D[k * N + j];
            resAD += A[i * N + k] * D[k * N + j];
            resBC += B[i * N + k] * C[k * N + j];
        }

        E[i*N + j] = resAC - resBD;
        F[i*N + j] = resAD + resBC;
    }
}

//========================================================================================================

// Kernel multipling matrices
__global__ void multiply_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && j < N)
    {
        for (int k = 0; k < N; k++)
            R[i * N + j] += M1[i * N + k] * M2[k * N + j];
    }
}

//========================================================================================================

// Kernel adding matrices
__global__ void add_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] + M2[pos];
    }
}

//========================================================================================================

// Kernel subtracting matrices
__global__ void sub_matrix(float *R, float *M1, float *M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] - M2[pos];
    }
}

//========================================================================================================

int main(int argc, char **argv)
{
    uint8_t MODE_CPU = 1<<0;
    uint8_t MODE_GPU_MULTIPLE = 1<<1; //Test multiple kernel implementation
    uint8_t MODE_GPU_SINGLE = 1<<2; //Test single kernel implementation
    uint8_t mode = MODE_CPU | MODE_GPU_SINGLE;

    if (argc == 1)
    {
        printf("Give matrix size\n");
        return 0;
    }

    if (argc >= 3)
    {
        mode = atoi(argv[2]);

        if ((mode > 7) || (mode < 1))
        {
            printf("Invalid mode. Must be in 1-7\n");
            exit(1);
        }
    }

    int N = atoi(argv[1]);

    //Open shared memory from Python driver program
    //===========================================================================
    int fd;
    double* shmem = NULL;
    
    if (argc == 4) //4th argument will be the shared memory name
    {
        fd = shm_open(argv[3], O_RDWR, 0);

        if (fd == -1)
            {perror("Could not open shared memory"); exit(1);}

        shmem = (double*)mmap(NULL, 6*sizeof(double), PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);

        if (close(fd) == -1)
            {perror("Could not close file descriptor"); exit(1);}

        if (shmem == MAP_FAILED)
            {perror("mmap failure"); exit(1);}
    }

    // Initialize matrices in host
    //===========================================================================
    float *A, *B, *C, *D;
    double t = initialize_matrices(&A, &B, &C, &D, N);
    if (shmem != NULL) shmem[0] = t;

    int arraySize = N * N * sizeof(float);

    int blockSize = 16;
    dim3 block(blockSize, blockSize); // 16x16 = 256 threads per block. A multiple of 32, the warp size
    dim3 grid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    //Start CPU calculations
    //===========================================================================
    float *Ecpu, *Fcpu;

    posix_memalign((void**)&Ecpu, 32, arraySize);
    posix_memalign((void**)&Fcpu, 32, arraySize);

    if (mode & MODE_CPU)
    {
        t = cpu_calculation(A, B, C, D, N, Ecpu, Fcpu);
        if (shmem != NULL) shmem[1] = t;
    }

    //Initialize GPU memory
    //===========================================================================
    float *devA, *devB, *devC, *devD, *devE, *devF, *devAC, *devBD, *devAD, *devBC;

    hipMalloc(&devA, arraySize);
    hipMalloc(&devB, arraySize);
    hipMalloc(&devC, arraySize);
    hipMalloc(&devD, arraySize);
    hipMalloc(&devE, arraySize);
    hipMalloc(&devF, arraySize);

    hipMemcpy(devA, A, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devC, C, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, arraySize, hipMemcpyHostToDevice);

    if (mode & MODE_GPU_MULTIPLE)
    {
        hipMalloc(&devAC, arraySize);
        hipMalloc(&devBD, arraySize);
        hipMalloc(&devAD, arraySize);
        hipMalloc(&devBC, arraySize);

        hipMemset(devAC, 0, arraySize);
        hipMemset(devBD, 0, arraySize);
        hipMemset(devAD, 0, arraySize);
        hipMemset(devBC, 0, arraySize);
    }

    //Start GPU calculations
    //===========================================================================
    float* E = (float *)malloc(arraySize);
    float* F = (float *)malloc(arraySize);

    if ((mode & MODE_GPU_MULTIPLE) || (mode & MODE_GPU_SINGLE))
        printf("Performing GPU calculations...\n\n");

    //Start multiple kernel calculations
    if (mode & MODE_GPU_MULTIPLE)
    {
        printf("> Running version with multiple kernels...\n-----------------------------------------------------\n");

        gpu_time();

        multiply_matrix<<<grid, block>>>(devAC, devA, devC, N);
        multiply_matrix<<<grid, block>>>(devBD, devB, devD, N);
        multiply_matrix<<<grid, block>>>(devAD, devA, devD, N);
        multiply_matrix<<<grid, block>>>(devBC, devB, devC, N);

        sub_matrix<<<grid, block>>>(devE, devAC, devBD, N);
        add_matrix<<<grid, block>>>(devF, devAD, devBC, N);

        t = gpu_time();
        if (shmem != NULL) shmem[2] = t;

        hipFree(devAC);
        hipFree(devBD);
        hipFree(devAD);
        hipFree(devBC);

        if (mode & MODE_CPU)
        {
            //Compare results for multiple kernels
            hipMemcpy(E, devE, arraySize, hipMemcpyDeviceToHost);
            hipMemcpy(F, devF, arraySize, hipMemcpyDeviceToHost);
            t = matrix_comparison(Ecpu, Fcpu, E, F, N);
            if (shmem != NULL) shmem[3] = t;
        }
    }

    //Start single kernel calculations
    if (mode & MODE_GPU_SINGLE)
    {
        printf("> Running version with single kernel...\n-----------------------------------------------------\n");

        gpu_time();
        single_kernel_calculations<<<grid, block>>>(devA, devB, devC, devD, devE, devF, N);
        t = gpu_time();
        if (shmem != NULL) shmem[4] = t;

        if (mode & MODE_CPU)
        {
            //Compare results for single kernel
            hipMemcpy(E, devE, arraySize, hipMemcpyDeviceToHost);
            hipMemcpy(F, devF, arraySize, hipMemcpyDeviceToHost);
            t = matrix_comparison(Ecpu, Fcpu, E, F, N);
            if (shmem != NULL) shmem[5] = t;
        }
    }

    //Free memory
    //========================================================================================
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipFree(devD);

    free(A);
    free(B);
    free(C);
    free(D);
    free(E);
    free(F);
    free(Ecpu);
    free(Fcpu);

    if (shmem != NULL)
    {
        if (munmap(shmem, sizeof(double)) == -1)
            {perror("unmap failure"); exit(1);}
    }
}