#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <math.h>

//========================================================================================================

void alloc_2d(float*** A, int N)
{
    *A = (float **)malloc(N * sizeof(float *));

    for (int i = 0; i < N; i++)
    {
        (*A)[i] = (float*)malloc(N * sizeof(float));
    }
}

//========================================================================================================

void initialize_matrix(float **matrix, int N)
{
    srand(time(NULL) + 1000 * omp_get_thread_num());
    // printf("%d\n", omp_get_thread_num());

    *matrix = (float *)malloc(N * N * sizeof(float));

    for (int i = 0; i < N * N; i++)
    {
        (*matrix)[i] = (rand() / (float)RAND_MAX) * 10;
    }
}

//========================================================================================================

double get_wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec * 1e-6;
}

//========================================================================================================

void print_matrix(float *matrix, int N)
{

    printf("==================================================================\n");

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.02f", matrix[i * N + j]);

            if (j < N - 1)
            {
                printf("\t");
            }
        }
        printf("\n");
    }
}

void print_matrix_cpu(float** matrix, int N)
{
    printf("==================================================================\n");

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.02f", matrix[i][j]);

            if (j < N - 1)
            {
                printf("\t");
            }
        }
        printf("\n");
    }
}

//========================================================================================================

__global__ void multiply_matrix(float *R, float *M1, float *M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // printf("(%d, %d)\n", i, j);

    if (i < N && j < N)
    {
        for (int k = 0; k < N; k++)
            R[i * N + j] += M1[i * N + k] * M2[k * N + j];
    }
}

//========================================================================================================

__global__ void add_matrix(float *R, float *M1, float *M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] + M2[pos];
    }
}

//========================================================================================================

__global__ void sub_matrix(float *R, float *M1, float *M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = i * N + j;

    if (i < N && j < N)
    {
        R[pos] = M1[pos] - M2[pos];
    }
}

//========================================================================================================

void cpu_matrix_add(float **AB, float **CD, float **result ,int n)
{

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            result[i][j] = AB[i][j] + CD[i][j];
        }
    }
}

//========================================================================================================

void cpu_matrix_sub(float **AB, float **CD, float **result ,int n)
{

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            result[i][j] = AB[i][j] - CD[i][j];
        }
    }
}

//========================================================================================================

void cpu_matrix_mull(float *A, float *B, float** result, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            result[i][j] = 0;
            for (int t = 0; t < n; t++)
            {
                result[i][j] += A[i * n + t] * B[t * n + j];
            }
        }
    }
}

//========================================================================================================

void free_memMatrix(float **matrix, int n){
    for(int i=0;i<n;i++){
        free(matrix[i]);
    }

    free(matrix);
}

//========================================================================================================

void cpu_calculation(float *A, float *B, float *C, float *D, int n, float** E, float** F)
{

    float **AC, **BD, **AD, **BC;

    AC = (float **)malloc(n * sizeof(float *));
    BD = (float **)malloc(n * sizeof(float *));
    AD = (float **)malloc(n * sizeof(float *));
    BC = (float **)malloc(n * sizeof(float *));

    for (int i = 0; i < n; i++)
    {
        AC[i] = (float *)malloc(n * sizeof(float));
        BD[i] = (float *)malloc(n * sizeof(float));
        AD[i] = (float *)malloc(n * sizeof(float));
        BC[i] = (float *)malloc(n * sizeof(float));
    }

    #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            cpu_matrix_mull(A, C, AC, n);

            #pragma omp task
            cpu_matrix_mull(B, D, BD, n);

            #pragma omp task
            cpu_matrix_mull(A, D, AD, n);

            #pragma omp task
            cpu_matrix_mull(B, C, BC, n);
        }
    }

    #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            cpu_matrix_sub(AC, BD, E, n);

            #pragma omp task
            cpu_matrix_add(AD, BC, F, n);
        }
    }

     #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            free_memMatrix(AC, n);

            #pragma omp task
            free_memMatrix(BD, n);
            
            #pragma omp task
            free_memMatrix(AD, n);

            #pragma omp task
            free_memMatrix(BC, n);
        }
    }
}

//========================================================================================================

void matrix_comparison(float **cpuE, float **cpuF,float* gpuE, float* gpuF, int n){
    
    int error=0;
    double tolerance = 1e-0;
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            if (fabs(cpuE[i][j] - gpuE[i * n + j]) > tolerance || 
                fabs(cpuF[i][j] - gpuF[i * n + j]) > tolerance){
                error=1;
                break;
            }
        }
        if(error==1){
            break;
        }
    }

    if(error==0){
        printf("Successful comparison\n");
    }else{
        printf("Comparison failed\n");
    }
    
}

//========================================================================================================

int main(int argc, char **argv)
{
    if (argc == 1)
    {
        printf("Give matrix size\n");
        return 0;
    }

    int N = atoi(argv[1]);

    // Initialize matrices in host
    //===========================================================================

    float *A, *B, *C, *D;

    #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            initialize_matrix(&A, N);

            #pragma omp task
            initialize_matrix(&B, N);

            #pragma omp task
            initialize_matrix(&C, N);

            #pragma omp task
            initialize_matrix(&D, N);
        }
    }

    /*
    printf("\nA\n");
    print_matrix(A, N);
    printf("\nB\n");
    print_matrix(B, N);
    printf("\nC\n");
    print_matrix(C, N);
    printf("\nD\n");
    print_matrix(D, N);*/

    float *devA, *devB, *devC, *devD, *devAC, *devBD, *devAD, *devBC;

    int arraySize = N * N * sizeof(float);

    hipMalloc(&devA, arraySize);
    hipMalloc(&devB, arraySize);
    hipMalloc(&devC, arraySize);
    hipMalloc(&devD, arraySize);
    hipMalloc(&devAC, arraySize);
    hipMalloc(&devBD, arraySize);
    hipMalloc(&devAD, arraySize);
    hipMalloc(&devBC, arraySize);

    hipMemset(devAC, 0, arraySize);
    hipMemset(devBD, 0, arraySize);
    hipMemset(devAD, 0, arraySize);
    hipMemset(devBC, 0, arraySize);

    hipMemcpy(devA, A, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devC, C, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, arraySize, hipMemcpyHostToDevice);

    int blockSize = 16;
    dim3 block(blockSize, blockSize); // 16x16 = 256 threads per block. A multiple of 32, the warp size
    dim3 grid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    multiply_matrix<<<grid, block>>>(devAC, devA, devC, N);
    multiply_matrix<<<grid, block>>>(devBD, devB, devD, N);
    multiply_matrix<<<grid, block>>>(devAD, devA, devD, N);
    multiply_matrix<<<grid, block>>>(devBC, devB, devC, N);

    sub_matrix<<<grid, block>>>(devAC, devAC, devBD, N);
    add_matrix<<<grid, block>>>(devAD, devAD, devBC, N);


    hipDeviceSynchronize(); //!!!!!!!!!!!!!!!!!!!

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Total time for GPU calculations: %.03lfs\n", milliseconds/1000);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Get back final results

    float *E = (float *)malloc(arraySize);
    float *F = (float *)malloc(arraySize);

    hipMemcpy(E, devAC, arraySize, hipMemcpyDeviceToHost);
    hipMemcpy(F, devAD, arraySize, hipMemcpyDeviceToHost);

    //print_matrix(E, N);
    //print_matrix(F, N);

    // Verify results
    //-------------------------------------------------------------------------------
    float **Echeck, **Fcheck;
    alloc_2d(&Echeck, N);
    alloc_2d(&Fcheck, N);

    cpu_calculation(A, B, C, D, N, Echeck, Fcheck);

    //print_matrix_cpu(Echeck, N);
    //print_matrix_cpu(Fcheck, N);

    matrix_comparison(Echeck,Fcheck,E,F,N);

    // Free memory

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipFree(devD);
    hipFree(devAC);
    hipFree(devBD);
    hipFree(devAD);
    hipFree(devBC);

    free(A);
    free(B);
    free(C);
    free(D);

    free_memMatrix(Echeck, N);
    free_memMatrix(Fcheck, N);
}