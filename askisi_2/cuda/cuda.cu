#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

void initialize_matrix(float** matrix, int N)
{
    srand(time(NULL) + 1000*omp_get_thread_num());
    //printf("%d\n", omp_get_thread_num());

    *matrix = (float*)malloc(N*N*sizeof(float));

    for (int i = 0; i < N*N; i++)
    {
        (*matrix)[i] = (rand() / (float)RAND_MAX) * 10;
    }
}

void print_matrix(float* matrix, int N)
{

    printf("==================================================================\n");

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%.02f", matrix[i*N + j]);

            if (j < N-1)
            {
                printf("\t");
            }
        }
        printf("\n");
    }
}


__global__ void multiply_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    //printf("(%d, %d)\n", i, j);

    if (i < N && j < N)
    {
        for (int k = 0; k < N; k++)
            R[i*N + j] += M1[i*N + k]*M2[k*N + j];
    }
}

int main(int argc, char** argv)
{
    if (argc == 1)
    {
        printf("Give matrix size\n");
        return 0;
    }

    int N = atoi(argv[1]);

    //Initialize matrices in host
    //===========================================================================

    float *A, *B, *C, *D;

    #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            initialize_matrix(&A, N);

            #pragma omp task
            initialize_matrix(&B, N);

            #pragma omp task
            initialize_matrix(&C, N);

            #pragma omp task
            initialize_matrix(&D, N);
        }
    }

    /*
    printf("\nA\n");
    print_matrix(A, N);
    printf("\nB\n");
    print_matrix(B, N);
    printf("\nC\n");
    print_matrix(C, N);
    printf("\nD\n");
    print_matrix(D, N);*/

    float *devA, *devB, *devC, *devD, *devR;

    int arraySize = N*N*sizeof(float);

    hipMalloc(&devA, arraySize);
    hipMalloc(&devB, arraySize);
    hipMalloc(&devC, arraySize);
    hipMalloc(&devD, arraySize);
    hipMalloc(&devR, arraySize);
    hipMemset(devR, 0, arraySize);

    hipMemcpy(devA, A, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devC, C, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, arraySize, hipMemcpyHostToDevice);

    print_matrix(A, N);
    print_matrix(B, N);

    int blockSize = 16;
    dim3 block(blockSize, blockSize); //16x16 = 256 threads per block. A multiple of 32, the warp size
    dim3 grid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    multiply_matrix<<<grid, block>>>(devR, devA, devB, N);

    float* result = (float*)malloc(arraySize);
    hipMemcpy(result, devR, arraySize, hipMemcpyDeviceToHost);

    print_matrix(result, N);
}