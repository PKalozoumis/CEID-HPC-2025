#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

void initialize_matrix(float** matrix, int N)
{
    srand(time(NULL) + 1000*omp_get_thread_num());
    printf("%d\n", omp_get_thread_num());

    *matrix = (float*)malloc(N*N*sizeof(float));

    for (int i = 0; i < N*N; i++)
    {
        (*matrix)[i] = (rand() / (float)RAND_MAX) * 1000;
    }
}

void print_matrix(float* matrix, int N)
{
    for (int i = 0; i < N*N; i++)
    {
        printf("%f\t", matrix[i]);

        if ((i > 0 && i % N == 0))
            printf("\n");
    }
}


__global__ void multiply_matrix(float* R, float* M1, float* M2, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    


}

int main(int argc, char** argv)
{
    if (argc == 1)
    {
        printf("Give matrix size\n");
        return 0;
    }

    int N = atoi(argv[1]);

    //Initialize matrices in host
    //===========================================================================

    float *A, *B, *C, *D;

    #pragma omp parallel
    {
        #pragma omp single
        {
            #pragma omp task
            initialize_matrix(&A, N);

            #pragma omp task
            initialize_matrix(&B, N);

            #pragma omp task
            initialize_matrix(&C, N);

            #pragma omp task
            initialize_matrix(&D, N);
        }
    }

    /*
    printf("\nA\n");
    print_matrix(A, N);
    printf("\nB\n");
    print_matrix(B, N);
    printf("\nC\n");
    print_matrix(C, N);
    printf("\nD\n");
    print_matrix(D, N);*/

    float *devA, *devB, *devC, *devD;

    int arraySize = N*N*sizeof(float);

    hipMalloc(&devA, arraySize);
    hipMalloc(&devB, arraySize);
    hipMalloc(&devC, arraySize);
    hipMalloc(&devD, arraySize);

    hipMemcpy(devA, A, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devC, C, arraySize, hipMemcpyHostToDevice);
    hipMemcpy(devD, D, arraySize, hipMemcpyHostToDevice);
}